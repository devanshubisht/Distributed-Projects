#include "hip/hip_runtime.h"
#include <vector>

#include "defs.h"

#define THREADS_PER_BLOCK 1024

// Unroll + int instead of size_t + shared bool
// __global__ void matchFile(const uint8_t* file_data, int file_len, 
// 	const char* sig_data, int sig_size, bool* flag_array, int sig_idx) 
// {
// 	__shared__ int shared_flag;
// 	if (threadIdx.x == 0) shared_flag = false;
// 	__syncthreads();

//     int file_start_byte_idx = blockIdx.x * blockDim.x + threadIdx.x;

// 	bool local_flag = true;
// 	for (int char_idx = 0; char_idx < sig_size; char_idx+=2) {

// 		int curr_file_byte_idx = file_start_byte_idx + (char_idx / 2);

// 		if (curr_file_byte_idx >= file_len){
// 			local_flag = false;
//  			break;
// 		}
		
// 		uint8_t curr_file_byte = file_data[curr_file_byte_idx];

// 		char sig_data_val1 = sig_data[char_idx];
// 		char sig_data_val2 = sig_data[char_idx + 1];

// 		uint8_t curr_sig_char_val1 = sig_data_val1 >= 'a' 
// 							? sig_data_val1 - 'a' + 10 
// 							: sig_data_val1 - '0';
		
// 		uint8_t curr_sig_char_val2 = sig_data_val2 >= 'a' 
// 							? sig_data_val2 - 'a' + 10 
// 							: sig_data_val2 - '0';
		
// 		uint8_t curr_sig_byte = (curr_sig_char_val1 << 4) | curr_sig_char_val2;

// 		if (sig_data_val1 != '?' && curr_file_byte != curr_sig_byte) {
// 			local_flag = false;
// 			break;
// 		}
// 	}

// 	if (local_flag) atomicOr(&shared_flag, 1);

// 	__syncthreads();

// 	if (threadIdx.x == 0) flag_array[sig_idx] = flag_array[sig_idx] || shared_flag;
// }


// Unroll + int instead of size_t
// __global__ void matchFile(const uint8_t* file_data, int file_len, 
// 	const char* sig_data, int sig_size, bool* flag_array, int sig_idx) 
// {
//     int file_start_byte_idx = blockIdx.x * blockDim.x + threadIdx.x;

// 	for (int char_idx = 0; char_idx < sig_size; char_idx+=2) {

// 		int curr_file_byte_idx = file_start_byte_idx + (char_idx / 2);

// 		if (curr_file_byte_idx >= file_len){
//  			return;
// 		}
		
// 		uint8_t curr_file_byte = file_data[curr_file_byte_idx];

// 		char sig_data_val1 = sig_data[char_idx];
// 		char sig_data_val2 = sig_data[char_idx + 1];

// 		// Convert the current signature char we are checking to uint8
// 		uint8_t curr_sig_char_val1 = sig_data_val1 >= 'a' 
// 							? sig_data_val1 - 'a' + 10 
// 							: sig_data_val1 - '0';
		
// 		uint8_t curr_sig_char_val2 = sig_data_val2 >= 'a' 
// 							? sig_data_val2 - 'a' + 10 
// 							: sig_data_val2 - '0';
		
// 		uint8_t curr_sig_byte = (curr_sig_char_val1 << 4) | curr_sig_char_val2;

		
// 		if (sig_data_val1 != '?' && curr_file_byte != curr_sig_byte) {
// 			return;
// 		}
// 	}
// 	// Set flag to true if the signature matches
// 	flag_array[sig_idx] = true;
// }

// unroll
__global__ void matchFile(const uint8_t* file_data, int file_len, 
	const char* sig_data, int sig_size, bool* flag_array, int sig_idx) 
{
    int file_start_byte_idx = blockIdx.x * blockDim.x + threadIdx.x;

	for (int char_idx = 0; char_idx < sig_size; char_idx+=2) {

		int curr_file_byte_idx = file_start_byte_idx + (char_idx / 2);

		if (curr_file_byte_idx >= file_len){
 			return;
		}
		
		uint8_t curr_file_byte = file_data[curr_file_byte_idx];

		char sig_data_val1 = sig_data[char_idx];
		char sig_data_val2 = sig_data[char_idx + 1];

		// Convert the current signature char we are checking to uint8
		uint8_t curr_sig_char_val1 = sig_data_val1 >= 'a' 
							? sig_data_val1 - 'a' + 10 
							: sig_data_val1 - '0';
		
		uint8_t curr_sig_char_val2 = sig_data_val2 >= 'a' 
							? sig_data_val2 - 'a' + 10 
							: sig_data_val2 - '0';
		
		uint8_t curr_sig_byte = (curr_sig_char_val1 << 4) | curr_sig_char_val2;

		
		if (sig_data_val1 != '?' && curr_file_byte != curr_sig_byte) {
			return;
		}
	}
	// Set flag to true if the signature matches
	flag_array[sig_idx] = true;
}

// Original
// __global__ void matchFile(const uint8_t* file_data, size_t file_len, 
// 	const char* sig_data, size_t sig_size, bool* flag_array, size_t sig_idx) 
// {
//     size_t file_start_byte_idx = static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x;

// 	for (size_t char_idx = 0; char_idx < sig_size; char_idx++) {

// 		size_t curr_file_byte_idx = file_start_byte_idx + (char_idx / 2);

// 		if (curr_file_byte_idx >= file_len){
//  			return;
// 		}
		
// 		uint8_t curr_file_char_val = (file_data[curr_file_byte_idx] >> 4 * (1 - (char_idx % 2))) & 0x0F;

// 		char curr_sig_char = sig_data[char_idx];

// 		// Convert the current signature char we are checking to uint8
// 		uint8_t curr_sig_char_val = curr_sig_char >= 'a' 
// 							? curr_sig_char - 'a' + 10 
// 							: curr_sig_char - '0';

		
// 		if (curr_sig_char != '?' && curr_file_char_val != curr_sig_char_val) {
// 			return;
// 		}
// 	}
// 	// Set flag to true if the signature matches
// 	flag_array[sig_idx] = true;
// }


void runScanner(std::vector<Signature>& signatures, std::vector<InputFile>& inputs)
{
	hipDeviceProp_t prop;
	check_cuda_error(hipGetDeviceProperties(&prop, 0));

	fprintf(stderr, "cuda stats:\n");
	fprintf(stderr, "  # of SMs: %d\n", prop.multiProcessorCount);
	fprintf(stderr, "  global memory: %.2f MB\n", prop.totalGlobalMem / 1024.0 / 1024.0);
	fprintf(stderr, "  shared mem per block: %zu bytes\n", prop.sharedMemPerBlock);
	fprintf(stderr, "  constant mem: %zu bytes\n", prop.totalConstMem);
	fprintf(stderr, "  max threads per block: %d\n", prop.maxThreadsPerBlock);
	fprintf(stderr, "  constant memory: %zu bytes\n", prop.totalConstMem);

	// Create one stream for each input file
	std::vector<hipStream_t> streams(inputs.size());

	// Copy input files to device
	std::vector<uint8_t*> file_bufs(inputs.size());

    std::vector<char*> sig_bufs(signatures.size());
    check_cuda_error(hipMallocManaged(sig_bufs.data(), signatures.size() * sizeof(char*)));

    for (size_t i = 0; i < signatures.size(); i++) {
        // Allocate managed memory for each signature
        char* ptr = nullptr;
        check_cuda_error(hipMallocManaged(&ptr, signatures[i].size));

        // Copy data from signatures to managed memory
        std::memcpy(ptr, signatures[i].data, signatures[i].size);

        // Assign the managed memory pointer to sig_bufs
        sig_bufs[i] = ptr;
    }


	size_t total_flags = inputs.size() * signatures.size();
	bool* flat_flags_array = nullptr;
	check_cuda_error(hipMallocManaged(&flat_flags_array, total_flags * sizeof(bool)));


	for(size_t file_idx = 0; file_idx < inputs.size(); file_idx++) {
		hipStreamCreate(&streams[file_idx]);
		uint8_t* file_ptr = 0; 

		
		check_cuda_error(hipMallocManaged(&file_ptr, sizeof(uint8_t) * inputs[file_idx].size));
		std::memcpy(file_ptr, inputs[file_idx].data, sizeof(uint8_t) * inputs[file_idx].size);
		file_bufs[file_idx]=file_ptr;

		bool* device_flag_array = &flat_flags_array[file_idx * signatures.size()];


		for(size_t sig_idx = 0; sig_idx < signatures.size(); sig_idx++) {
			size_t NUM_WINDOWS = (inputs[file_idx].size - (signatures[sig_idx].size / 2)) + 1;
			size_t NUM_BLOCKS = (NUM_WINDOWS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

			// NORMAL and UNROLL
			matchFile<<<NUM_BLOCKS, THREADS_PER_BLOCK, 0, streams[file_idx]>>>(
				file_bufs[file_idx], 
				static_cast<int>(inputs[file_idx].size), 
				sig_bufs[sig_idx], 
				static_cast<int>(signatures[sig_idx].size),
				device_flag_array,
				static_cast<int>(sig_idx)
			);

			// SIZE_T to INT
			// matchFile<<<NUM_BLOCKS, THREADS_PER_BLOCK, 0, streams[file_idx]>>>(
			// 	file_bufs[file_idx], 
			// 	static_cast<int>(inputs[file_idx].size), 
			// 	sig_bufs[sig_idx], 
			// 	static_cast<int>(signatures[sig_idx].size),
			// 	device_flag_arrays[file_idx],
			// 	static_cast<int>(sig_idx)
			// );

			// SIZE_T to INT + SHARED MEM
			// matchFile<<<NUM_BLOCKS, THREADS_PER_BLOCK, sizeof(int), streams[file_idx]>>>(
			// 	file_bufs[file_idx], 
			// 	static_cast<int>(inputs[file_idx].size), 
			// 	sig_bufs[sig_idx], 
			// 	static_cast<int>(signatures[sig_idx].size),
			// 	device_flag_arrays[file_idx],
			// 	static_cast<int>(sig_idx)
			// );
		}
	}

	// Synchronize streams
	for (size_t file_idx = 0; file_idx < inputs.size(); file_idx++) {
		hipStreamSynchronize(streams[file_idx]);
		for (size_t sig_idx = 0; sig_idx < signatures.size(); sig_idx++) {
			if (flat_flags_array[file_idx * signatures.size() + sig_idx]) {
				printf("%s: %s\n", inputs[file_idx].name.c_str(), signatures[sig_idx].name.c_str());
			}
		}
		hipFree(file_bufs[file_idx]);
	}

	// free the device memory, though this is not strictly necessary
	// (the CUDA driver will clean up when your program exits)
    
    for (size_t i = 0; i < signatures.size(); i++) {
        hipFree(sig_bufs[i]); // Free individual signature buffers
    }

    hipFree(sig_bufs.data()); // Free the managed memory for sig_bufs

	// clean up streams (again, not strictly necessary)
	for(auto& s : streams)
		hipStreamDestroy(s);
}
